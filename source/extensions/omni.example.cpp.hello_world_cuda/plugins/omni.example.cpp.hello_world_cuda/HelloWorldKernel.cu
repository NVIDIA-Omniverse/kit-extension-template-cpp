#include <hip/hip_runtime.h>
#include <cstdio>

// A very small CUDA kernel that prints from the device.
__global__ void helloKernel(void)
{
    printf("Hello from CUDA kernel! (block %d, thread %d)\n", blockIdx.x, threadIdx.x);
}

// Launch the kernel and wait for it to complete.
extern "C" bool launchCudaHelloKernel(void)
{
    // Launch one block with four threads.
    helloKernel<<<1, 4>>>();
    hipError_t err = hipDeviceSynchronize();
    return err == hipSuccess;
}